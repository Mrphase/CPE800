#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <algorithm>
//#include "graph.h"
#include <string>
#include <iostream>
#include <chrono>
#include <string>
#include <list>

#include <stdio.h>
#include <> //.显示blockDim等变量出现了未定义的错误：
#include <hip/hip_runtime.h>
#include <assert.h>
#include <./inc/hip/hip_runtime_api.h> //C:\ProgramData\NVIDIA Corporation\CUDA Samples\v10.1\common\inc\hip/hip_runtime_api.h
//#include<C:\ProgramData\NVIDIA Corporation\CUDA Samples\v10.1\common\inc\helper_string.h>

#include <hipsparse.h>

//#include "../common/common.h"

#include <stdio.h>

#include <stdlib.h>

#include <hipsparse.h>

#include <hip/hip_runtime.h>


#define blockMatrixSize	
// #include <ext/hash_map> //gcc ?
// #include <hash_map>
#include <unordered_map> 

using namespace std::chrono;
using namespace std;
//using namespace __gnu_cxx;
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void initialize(float* cooValHostPtr, int* cooColIndexHostPtr, float* yHostPtr, int* csrRowPtr)
{
	cooValHostPtr[0] = 1.0;
	cooValHostPtr[1] = 2.0;
	cooValHostPtr[2] = 3.0;
	cooValHostPtr[3] = 4.0;
	cooValHostPtr[4] = 5.0;
	cooValHostPtr[5] = 6.0;
	cooValHostPtr[6] = 7.0;
	cooValHostPtr[7] = 8.0;
	cooValHostPtr[8] = 9.0;

	cooValHostPtr[9] = 10.0;

	cooColIndexHostPtr[0] = 0;
	cooColIndexHostPtr[1] = 2;
	cooColIndexHostPtr[2] = 3;
	cooColIndexHostPtr[3] = 1;
	cooColIndexHostPtr[4] = 0;
	cooColIndexHostPtr[5] = 2;
	cooColIndexHostPtr[6] = 3;
	cooColIndexHostPtr[7] = 1;
	cooColIndexHostPtr[8] = 3;

	cooColIndexHostPtr[9] = 0;

	yHostPtr[0] = 10.0;
	yHostPtr[1] = 20.0;
	yHostPtr[2] = 30.0;
	yHostPtr[3] = 40.0;
	yHostPtr[4] = 50.0;
	yHostPtr[5] = 60.0;
	yHostPtr[6] = 70.0;
	yHostPtr[7] = 80.0;

	csrRowPtr[0] = 0;
	csrRowPtr[1] = 3;
	csrRowPtr[2] = 4;
	csrRowPtr[3] = 7;
	csrRowPtr[4] = 9;

	csrRowPtr[5] = 10;

}

void cuda_sparse()
{
	int m = 5, n = 4, nnz = 10;
	float* cooValHostPtr = new float[nnz];
	float* zHostPtr = new float[2 * (m)];

	int* cooColIndexHostPtr = new int[nnz];
	int* csrRowPtr = new int[m + 1];

	int* crsRow, * cooCol;

	float alpha = 1;
	float beta = 0;
	float* yHostPtr = new float[2 * n];
	float* y, * cooVal, * z;
	initialize(cooValHostPtr, cooColIndexHostPtr, yHostPtr, csrRowPtr);


	hipsparseHandle_t handle;
	hipsparseMatDescr_t descr;
	(hipsparseCreateMatDescr(&descr));
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	(hipsparseCreate(&handle));

	(hipMalloc((void**)&cooVal, nnz * sizeof(float)));
	(hipMalloc((void**)&y, 2 * n * sizeof(float)));
	(hipMalloc((void**)&z, 2 * (m) * sizeof(float)));
	(hipMalloc((void**)&crsRow, (m + 1) * sizeof(int)));
	(hipMalloc((void**)&cooCol, nnz * sizeof(int)));

	hipMemcpy(cooVal, cooValHostPtr, nnz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y, yHostPtr, 2 * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(crsRow, csrRowPtr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cooCol, cooColIndexHostPtr, nnz * sizeof(int), hipMemcpyHostToDevice);

	(hipMemset((void*)z, 0, 2 * (m) * sizeof(float)));

	(hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, 2, n, nnz, &alpha, descr, cooVal, crsRow, cooCol, y, n, &beta, z, m));

	(hipMemcpy(zHostPtr, z, 2 * (m) * sizeof(float), hipMemcpyDeviceToHost));

	//for (int i = 0; i < m; i++)
	//{
	//  //if(i%(2)==0&&i!=0)
	//  //  cout<<endl;
	//  cout<<zHostPtr[i]<<" "<<zHostPtr[i+m]<<endl;
	//}
	for (int i = 0; i < m * 2; i++)
	{
		cout << zHostPtr[i] << " ";
	}
}

int main()
{
	cuda_sparse();
	return 0;
}
//
//
//
//int main()
//{
//	cout << "hds";
//    return 0;
//}
