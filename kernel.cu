#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <mma.h>

using namespace nvcuda;

__global__ void wmma_ker(half* a, half* b, float* c) {
	// Declare the fragments
	wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag;
	wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
	wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

	// Initialize the output to zero
	wmma::fill_fragment(c_frag, 0.0f);

	// Load the inputs
	wmma::load_matrix_sync(a_frag, a, 16);
	wmma::load_matrix_sync(b_frag, b, 16);

	// Perform the matrix multiplication
	wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

	// Store the output
	wmma::store_matrix_sync(c, c_frag, 16, wmma::mem_row_major);
}
// Create a cuDNN handle:
checkCudnnErr(hipdnnCreate(&handle_));

// Create your tensor descriptors:
checkCudnnErr( hipdnnCreateTensorDescriptor( &cudnnIdesc ));
checkCudnnErr( hipdnnCreateFilterDescriptor( &cudnnFdesc ));
checkCudnnErr( hipdnnCreateTensorDescriptor( &cudnnOdesc ));
checkCudnnErr( hipdnnCreateConvolutionDescriptor( &cudnnConvDesc ));

// Set tensor dimensions as multiples of eight (only the input tensor is shown here):
int dimA[] = {1, 8, 32, 32};
int strideA[] = {8192, 1024, 32, 1};

checkCudnnErr( hipdnnSetTensorNdDescriptor(cudnnIdesc, getDataType(), 
                                          convDim+2, dimA, strideA) );

// Allocate and initialize tensors (again, only the input tensor is shown):
checkCudaErr( hipMalloc((void**)&(devPtrI), (insize) * sizeof(devPtrI[0]) ));
hostI = (T_ELEM*)calloc (insize, sizeof(hostI[0]) );

initImage(hostI, insize);

checkCudaErr( hipMemcpy(devPtrI, hostI, sizeof(hostI[0]) * insize, hipMemcpyHostToDevice));

// Set the compute data type (below as HIPDNN_DATA_FLOAT):
checkCudnnErr( hipdnnSetConvolutionNdDescriptor(cudnnConvDesc,
                                               convDim,
                                               padA,
                                               convstrideA,
                                               dilationA,
                                               HIPDNN_CONVOLUTION,
                                               HIPDNN_DATA_FLOAT) );

// Set the math type to allow cuDNN to use Tensor Cores:
checkCudnnErr( hipdnnSetConvolutionMathType(cudnnConvDesc, HIPDNN_TENSOR_OP_MATH) );

// Choose a supported algorithm:
hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

// Allocate your workspace:
checkCudnnErr( hipdnnGetConvolutionForwardWorkspaceSize(handle_, cudnnIdesc, 
                                                       cudnnFdesc, cudnnConvDesc,
                                                       cudnnOdesc, algo, &workSpaceSize) );

if (workSpaceSize > 0) {
   hipMalloc(&workSpace, workSpaceSize);
}

// Invoke the convolution:
checkCudnnErr( hipdnnConvolutionForward(handle_, (void*)(&alpha), cudnnIdesc, devPtrI,
                                       cudnnFdesc, devPtrF, cudnnConvDesc, algo,
                                       workSpace, workSpaceSize, (void*)(&beta),
                                       cudnnOdesc, devPtrO) );
