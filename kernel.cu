
#include "hip/hip_runtime.h"


#include <stdio.h>

#include <mma.h>

using namespace nvcuda;

__global__ void wmma_ker(half* a, half* b, float* c) {
	// Declare the fragments
	wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag;
	wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
	wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;

	// Initialize the output to zero
	wmma::fill_fragment(c_frag, 0.0f);

	// Load the inputs
	wmma::load_matrix_sync(a_frag, a, 16);
	wmma::load_matrix_sync(b_frag, b, 16);

	// Perform the matrix multiplication
	wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

	// Store the output
	wmma::store_matrix_sync(c, c_frag, 16, wmma::mem_row_major);
}