#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "../CPE800_CUDA_APSP/apsp_misc.h"
#include "../CPE800_CUDA_APSP/apsp_parallel_1.h"
#include "../CPE800_CUDA_APSP/apsp_misc.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <string>
#include <iostream>
#include <fstream>
using namespace std::chrono;
using namespace std;
#define NN  999
const int smp_executions = 8192;
const int threads_per_block = 128;
const int threads_per_smp = 2048;

// derived 
const int blocks_per_smp = threads_per_smp / threads_per_block;
const dim3 blocks(smp_executions, blocks_per_smp);
const dim3 threads(threads_per_block);

__global__ void apsp_parallel_1_kernel(float* dev_dist, int N, int k) {

	int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	int i, j;
	float dist1, dist2, dist3;

	if (tid < N * N) {

		i = tid / N;
		j = tid - i * N;

		dist1 = dev_dist[tid];
		dist2 = dev_dist[i * N + k];
		dist3 = dev_dist[k * N + j];

		if (dist1 > dist2 + dist3)
			dev_dist[tid] = dist2 + dist3;
	}
}

// CUDA of Floyd Warshall algorithm
void apsp_parallel_1(float** graph, float** dist, int N) {


	float* dev_dist;
	hipMalloc((void**)&dev_dist, N * N * sizeof(float));

	
	for (int i = 0; i < N; i++)
		hipMemcpy(dev_dist + i * N, graph[i], N * sizeof(float),
			hipMemcpyHostToDevice);

	
	for (int k = 0; k < N; k++) {

		// launch kernel
		apsp_parallel_1_kernel << <blocks, threads >> > (dev_dist, N, k);
		
	}

	// return results to dist matrix on host
	for (int i = 0; i < N; i++)
		hipMemcpy(dist[i], dev_dist + i * N, N * sizeof(float),
			hipMemcpyDeviceToHost);
	hipFree((void**)&dev_dist);
}


int main()
{
	int row = 1024* 2, col = 1024*2; // col and row
	int INF = 9999;

	cout << "Normal 65!!! ";
	typedef struct
	{
		//结构体
		int row, col;
		//二维指针，目的是动态分配内存
		float** matrix;
	} Matrix;
	cout << "Normal 73!!! ";
	typedef struct
	{
		int row, col;
		string** matrix;
	} Matrix_string;

	Matrix m;         //store value of path
	Matrix_string m2; //store path
	std::cout << "Normal 91!!!put in to memory "
		<< "\n";

	float** enterMatrix;
	string** enterMatrix2;
	enterMatrix = (float**)malloc(row * sizeof(float*));   // value of path
	enterMatrix2 = (string**)malloc(row * sizeof(float*)); //store path
	std::cout << "Normal 103!!! "
		<< "\n";
	for (int i = 0; i < row; i++) //put in to memory  //change size to *10
	{
		enterMatrix[i] = (float*)malloc(col * 10 * sizeof(float));
		enterMatrix2[i] = (string*)malloc(col * 10 * sizeof(string)); // change sizeof(float) to string
	}
	std::cout << "Normal 109!!! set default value"
		<< "\n";
	int count_of_nuZero = 0;
	for (int i = 0; i < row; i++) //set default value
	{
		for (int j = 0; j < col; j++)
		{
			enterMatrix[i][j] = INF;  //For path value ,default is 99999
			//enterMatrix2[i][j] = " "; //for path, default is " "
			if (i == j )
			{
				enterMatrix[i][j] = 0;
			}
			if (rand() % (NN + 1) / (float)(NN + 1)>0.9)
			{
				enterMatrix[i][j] = 1;
				count_of_nuZero++;
			}

		}
		
	}
	std::cout << "Normal 128!!! GPU start"
		<< "\n";
	//print_array(arr, n);
	auto start = high_resolution_clock::now();	
	apsp_parallel_1(enterMatrix, enterMatrix, row);
	auto stop = high_resolution_clock::now();
	//print_array(arr, n);
	auto duration = duration_cast<microseconds>(stop - start);
	cout << "apsp_parallel_1  1024     " << duration.count() << " ms   " << "\n";


	std::cout << "Normal 140!!! Please check your algrothm using below demo"
		<< "\n";

	float** M;
	M = (float**)malloc(100 * sizeof(float*));
	for (int i = 0; i < 100; i++) //put in to memory  //change size to *10
	{
		M[i] = (float*)malloc(col * 10 * sizeof(float));
	}
	count_of_nuZero = 0;
	for (int i = 0; i < 10; i++) //set default value
	{
		for (int j = 0; j < 10; j++)
		{
			M[i][j] = 99;  //For path value ,default is 99999
			//enterMatrix2[i][j] = " "; //for path, default is " "
			if (rand() % 100>70)
			{
				M[i][j] = 1;
				count_of_nuZero++;
			}
			if (i==j)
			{
				M[i][j] = 0;
			}
			std::cout << M[i][j] << " ";

		}
		std::cout 
		<< "\n";
	}
	std::cout << count_of_nuZero
		<< "\n";

	std::cout<< "M complete" << "\n";
	float** M2 = M;

	apsp_parallel_1(M, M2, 10);

	auto start2 = high_resolution_clock::now();
	apsp_parallel_1(M, M2, 10);
	auto stop2 = high_resolution_clock::now();
	auto duration2 = duration_cast<microseconds>(stop2 - start2);
	cout << "apsp_parallel_1  10     " << duration2.count() << " ms   "<<"\n";


	for (int i = 0; i < 10; i++) //set default value
	{
		for (int j = 0; j < 10; j++)
		{
			std::cout << M[i][j]<<" ";
		}
		std::cout
			<< "\n";
	}

	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	std::cout << "//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////"
		<< "\n";
	std::cout << "Normal 200!!! start to test running time from 32 to 65536"
		<< "\n";
	ofstream fout("E:\\CPE800_CUDA_APSP\\mytest.txt");
	for (int ii = 32; ii < 32769; ii*=2)
	{

		int row = ii, col = ii; // col and row
		int INF = 9999;

		cout << "Normal 65!!! ";
		typedef struct
		{
			//结构体
			int row, col;
			//二维指针，目的是动态分配内存
			float** matrix;
		} Matrix;
		cout << "Normal 73!!! ";
		typedef struct
		{
			int row, col;
			string** matrix;
		} Matrix_string;

		Matrix m;         //store value of path
		Matrix_string m2; //store path
		std::cout << "Normal 91!!!put in to memory "
			<< "\n";

		float** enterMatrix;
		string** enterMatrix2;
		enterMatrix = (float**)malloc(row * sizeof(float*));   // value of path
		enterMatrix2 = (string**)malloc(row * sizeof(float*)); //store path
		std::cout << "Normal 103!!! "
			<< "\n";
		for (int i = 0; i < row; i++) //put in to memory  //change size to *10
		{
			enterMatrix[i] = (float*)malloc(col * 10 * sizeof(float));
			enterMatrix2[i] = (string*)malloc(col * 10 * sizeof(string)); // change sizeof(float) to string
		}
		std::cout << "Normal 109!!! set default value"
			<< "\n";
		int count_of_nuZero = 0;
		for (int i = 0; i < row; i++) //set default value
		{
			for (int j = 0; j < col; j++)
			{
				enterMatrix[i][j] = INF;  //For path value ,default is 99999
				//enterMatrix2[i][j] = " "; //for path, default is " "
				if (i == j)
				{
					enterMatrix[i][j] = 0;
				}
				if (rand() % (NN + 1) / (float)(NN + 1) > 0.95)
				{
					enterMatrix[i][j] = 1;
					count_of_nuZero++;
				}

			}

		}
		std::cout << "Normal 128!!! GPU start"
			<< "\n";
		//print_array(arr, n);
		auto start = high_resolution_clock::now();
		apsp_parallel_1(enterMatrix, enterMatrix, row);
		auto stop = high_resolution_clock::now();
		//print_array(arr, n);
		auto duration = duration_cast<microseconds>(stop - start);
		cout << "apsp_parallel_1  edge：    "<< count_of_nuZero << "  node:  "<<ii<<"  " << duration.count() << " ms   " << "\n";
		
		fout << count_of_nuZero << "   "<< duration.count() << " ms   " << "\n";
		//fout.close();
	}

	return 0;
}
